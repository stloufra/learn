#include "hip/hip_runtime.h"
#include <iostream>
#include <iostream>
#include <chrono>
#include <Eigen/Dense>
#include <hip/hip_cooperative_groups.h>

#define NUM_OF_MAT 120
#define MATRIX_PRINT 49

// Function to check CUDA errors
#define CHECK_CUDA(call) \
        do { \
            hipError_t error = call; \
            if (error != hipSuccess) { \
                std::cerr << "CUDA error: " << hipGetErrorString(error) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
                exit(EXIT_FAILURE); \
            } \
        } while (0)

template<typename T>
__host__ __device__ void fillMatrixVal(T &A, double a) {

    auto N = T::ColsAtCompileTime;

    for (int j = 0; j < N; j++) {
        for (int i = 0; i < N; i++) {
            A(i, j) = ((double)j+0.523f)*(13.f/18.f)*((double)i+0.87f) + a/25.13f;
        }
    }
}


template<typename MATRIX, typename TILE>
__device__ inline void ABAteqC(MATRIX const &A, MATRIX const &B, MATRIX &C, MATRIX &holder, TILE &tile) {

    auto N = MATRIX::ColsAtCompileTime;

    auto idx = tile.thread_rank();
    double tmp;
    unsigned int row;
    unsigned int col;

    for (auto i = idx; i < N * N; i += tile.num_threads()) {
        row = i / N;
        col = i % N;
        tmp = 0;

        for (unsigned int m = 0; m < N; m++) {
            for (unsigned int k = 0; k < N; k++) {
                tmp += A(row, k) * B(k, m) * A(col, m);
            }
        }
        holder(row, col) = tmp;
    }
    tile.sync();
    C = holder.eval();
    tile.sync();
}

template <typename MATRIX, typename TILE>
__device__ inline void jacobiMult(MATRIX const& A, MATRIX const& B, MATRIX& C, MATRIX& holder, TILE& tile) {
    ABAteqC(A, B, C, holder, tile);
}

template<typename MATRIX, typename TILE, typename NAME>
__device__ void printFromDevice(MATRIX A, TILE tile, NAME name) {
    auto const N = MATRIX::ColsAtCompileTime;
    auto idx = tile.thread_rank();

    if (idx == 0 && tile.meta_group_rank() == 0) {
        if (N == 3) {
            printf("MATRIX %s IS:{[%f,%f,%f],[%f,%f,%f],[%f,%f,%f]\n}", name, A(0,0), A(0,1), A(0,2), A(1,0), A(1,1), A(1,2), A(2,0), A(2,1), A(2,2));
        }
        if (N == 2) {
            printf("MATRIX %s IS:{[%f,%f],[%f,%f]\n}", name, A(0,0), A(0,1), A(1,0), A(1,1));
        }
    }

}

template<typename MATRIX>
__global__ void myKERNEL(MATRIX A,
                         MATRIX B,
                         MATRIX C) {


    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<4> tile = cg::tiled_partition<4>(block);

    auto tileId = tile.meta_group_rank();


    printFromDevice(A[MATRIX_PRINT], tile, "A");
    block.sync();
    printFromDevice(B[MATRIX_PRINT], tile, "B");
    block.sync();
    printFromDevice(C[MATRIX_PRINT], tile, "C");

    __shared__ Eigen::Matrix2d A_shr[NUM_OF_MAT];
    __shared__ Eigen::Matrix2d B_shr[NUM_OF_MAT];
    __shared__ Eigen::Matrix2d holder[NUM_OF_MAT];

    for(auto idx = tileId; idx < NUM_OF_MAT; idx+=tile.meta_group_size()){
        A_shr[idx] = A[idx];
        B_shr[idx] = B[idx];
        jacobiMult(A_shr[idx], B_shr[idx], B_shr[idx], holder[idx], tile);
        B[idx] = B_shr[idx];
    }

    block.sync();

    printFromDevice(B[MATRIX_PRINT], tile, "C after");
    if(block.thread_rank()==0) printf("\n");
    block.sync();
}

int main() {

    using Matrix = Eigen::Matrix2d;


    Matrix A[NUM_OF_MAT];
    Matrix B[NUM_OF_MAT];
    Matrix C[NUM_OF_MAT];
    Matrix Cmy[NUM_OF_MAT];

    Matrix Ad[NUM_OF_MAT];
    Matrix Bd[NUM_OF_MAT];
    Matrix Cd[NUM_OF_MAT];

    size_t num_bytes = sizeof(Matrix)*NUM_OF_MAT;
    size_t num_bytes_copy = sizeof(double)*Matrix::ColsAtCompileTime*Matrix::ColsAtCompileTime;

    CHECK_CUDA(hipMalloc((void **) &Ad, num_bytes));
    CHECK_CUDA(hipMalloc((void **) &Bd, num_bytes));
    CHECK_CUDA(hipMalloc((void **) &Cd, num_bytes));

    for(int i = 0; i < NUM_OF_MAT; i++) {
        fillMatrixVal(A[i], (double)i);
        fillMatrixVal(B[i], (double)i);
        CHECK_CUDA(hipMemcpy(Ad[i].data(), A[i].data(), num_bytes_copy, hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(Bd[i].data(), B[i].data(), num_bytes_copy, hipMemcpyHostToDevice));
        C[i] = A[i] * B[i] * A[i].transpose();
    }

    std::cout << A[MATRIX_PRINT] << "\nX\n" << B[MATRIX_PRINT] << "\nX\n" << A[MATRIX_PRINT].transpose() << "\n=\n" << C[MATRIX_PRINT] << "\n\n";


    myKERNEL<<<1, 32>>>(Ad, Bd, Cd);

    CHECK_CUDA(hipGetLastError());
    hipDeviceSynchronize();

    for(int i = 0; i < NUM_OF_MAT; i++) {
        CHECK_CUDA(hipMemcpy(Cmy[i].data(), Bd[i].data(), num_bytes_copy, hipMemcpyDeviceToHost));
    }

    std::cout << A[MATRIX_PRINT] << "\nX\n" << B[MATRIX_PRINT] << "\nX\n" << A[MATRIX_PRINT].transpose() << "\n=\n" << Cmy[MATRIX_PRINT] << "\n\n";

    for(int i = 0; i < NUM_OF_MAT; i++){
        auto diff = (C[i]-Cmy[i]);
        auto err = diff.norm();
        if(err > 0.00001) printf("HERE I AM WRONG! %d\n", i);
    }

    hipFree((void **) &Ad);
    hipFree((void **) &Bd);
    hipFree((void **) &Cd);
    return 0;


}