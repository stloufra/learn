#include <hip/hip_runtime.h>

#include <iostream>
#include <cassert>

//Threads       - executes the instruction
//Warps         - SIMT (lowest schedulable entity) size 32
//Thread blocks - lowest programmable entity
//              - assignet to shader core
//Grids         - mapping to gpu

//    T(x,y),B(x,y)            0        1    .   2       3
//    0                    0,0;0,0           .
//    1                                      .
//      .....................................................................................
//    2                                      .0,0;1,1
//    3                                      .

//Row = blockIdx.y*blockDim.y + threadIdx.y
//Col = blockIdx.x*blockDim.x + threadIdx.x

//scratch memory | user managed L1 cache | private per-thread block

using realType = int;

#define SHMEM_SIZE 16*16*sizeof(realType)


void matrixInit(realType *a, int n) {

    for (int i = 0; i < n; i++) { //radky
        for (int j = 0; j < n; j++) { //sloupce
            a[i * n + j] = (realType) j;
        }
    }
}

void matrixMultHost(realType *a, realType *b, realType *ab, int N) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            for (int R = 0; R < N; R++) {
                ab[i * N + j] += a[i * N + R] * b[R * N + j];
            }
        }
    }
}

void printMatix(realType *A, int N) {

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (j != N - 1) {
                printf("%d, ", A[i * N + j]);
            } else {
                printf("%d", A[i * N + j]);
            }
        }
        printf("\n");
    }
    printf("\n");
}

void checkIfSame(realType *a, realType *b, int N) {


    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            assert(a[i * N + j] == b[i * N + j]);
#ifdef UNMUTED
            printf("%d  = %d \n", a[i * N + j], b[i * N + j] );
#endif
        }
    }


    std::cout << "IT IS GREAT SUCCESS!" << std::endl;
}

__global__ void matrixMult(realType *a, realType *b, realType *c, int N) {


    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;


    realType tempSum = 0;
    if ((row < N) && (col < N)) {

        for (int k = 0; k < N; k++) {
            tempSum += a[row * N + k] * b[k * N + col];
        }
    }

    c[row * N + col] = tempSum;
}

__global__ void matrixMultShared(realType *a, realType *b, realType *c, int N, int tile_size) {


    __shared__ realType A[SHMEM_SIZE];
    __shared__ realType B[SHMEM_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;


    int row = by * tile_size + ty; //global
    int col = bx * tile_size + tx;

    realType tempSum = 0;

    //every thread in threadblock loads one element in shared memory
    //thread [0,0] -> A[0*tile_size + 0]
    // A:
    //      row*n -> global row for thread (loop-inv)
    //      i*tile_size -> new set of colum for each iteration
    //      tx -> colum within that set
    // B:
    //      i*tile_size -> new set of rows within the set
    //      ty*n -> row within that set
    //      col -> colum global (loop-inv)

    for (int i = 0; i < (N / tile_size); i++) {

        A[(ty * tile_size) + tx] = a[row * N + (i * tile_size + tx)];
        B[(ty * tile_size) + tx] = b[(i * tile_size + ty) * N + col];

        __syncthreads();


        for (int k = 0; k < tile_size; k++) {
            tempSum += A[ty * tile_size + k] * b[k * tile_size + tx];
        }

        __syncthreads();

    }

    c[row * N + col] = tempSum;
}


int main() {


    int N = 3 << 10; // shift operator x * (2^y)


    size_t bytes = sizeof(realType) * N * N;

    printf("Our size in bites %d \n", bytes);

    int BLOCK_SIZE = 16; //16*16=256
    int GRID_SIZE = (int) ceil((float) N / BLOCK_SIZE); // multiple of 32

    std::cout << "GRID_SIZE: " << GRID_SIZE << std::endl;

    dim3 grid(GRID_SIZE, GRID_SIZE);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);

    //-----------------
    // managed memory
    //-----------------
    realType *h_a, *h_b, *h_c, *h_d;
    realType *d_a, *d_b, *d_c;

    //Alocate host memory
    h_a = (realType *) malloc(bytes);
    h_b = (realType *) malloc(bytes);
    h_c = (realType *) malloc(bytes);
    h_d = (realType *) malloc(bytes);

    //Allocate CUDA memory
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    matrixInit(h_a, N);
    matrixInit(h_b, N);


    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    //-----------------------------------------------------------------------------------
    clock_t startNorm = clock();
    matrixMult <<<grid, block>>>(d_a, d_b, d_c, N);
    clock_t endNorm = clock();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();

    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    //-----------------------------------------------------------------------------------
    clock_t startShared = clock();
    matrixMultShared <<<grid, block>>>(d_a, d_b, d_c, N, 16);
    clock_t endShared = clock();

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

    hipMemcpy(h_d, d_c, bytes, hipMemcpyDeviceToHost);

    checkIfSame(h_d, h_c, N);

    printf("Normal kernel needed %f vs %f for shared\n", (float)(-startNorm+endNorm)/CLOCKS_PER_SEC*1e6, (float)(-startShared+endShared)/CLOCKS_PER_SEC*1e6 );

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_d);
    free(h_c);
    free(h_b);
    free(h_a);


    return 0;
}

